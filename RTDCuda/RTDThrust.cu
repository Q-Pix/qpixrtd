#include "RTDThrust.h"

#include <cstdlib>
#include <chrono>

extern "C" void Launch_ThrustSort(unsigned int* a, int size)
{
    unsigned int* d_vals;
    hipMalloc((void**)&d_vals, size * sizeof(unsigned int));
    hipMemcpy(d_vals, a, size * sizeof(unsigned int), hipMemcpyHostToDevice);

    // kernel launch
    ThrustSort(a, d_vals, size);
}

void ThrustSort(unsigned int* h_a, unsigned int* d_a, int size)
{
    /* 5613 us for 13e6 keys */
    // auto start = std::chrono::high_resolution_clock::now();
    thrust::device_ptr<unsigned int> td_vals(d_a);
    thrust::sort(td_vals, td_vals+size);
    // auto stop = std::chrono::high_resolution_clock::now();
    // auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
    // std::cout << "launch thrust call time: " << duration.count() << "\n";

    /* 5197 us for 13e6 keys */
    // start = std::chrono::high_resolution_clock::now();
    hipMemcpy(h_a, d_a, size * sizeof(unsigned int), hipMemcpyDeviceToHost);
    // stop = std::chrono::high_resolution_clock::now();
    // duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
    // std::cout << "thrust copy time: " << duration.count() << "\n";

    // print the sorted result
    // if(err != 0)std::cout << "err.\n";
    // std::cout << "out keys: { ";
    // for(int i=0; i<size; i++) std::cout << h_a[i] << " ";
    // std::cout << "}\n";
}

extern "C" void Launch_ThrustSortStruct(pix* a, int size)
{
    pix* d_vals;
    hipMalloc((void**)&d_vals, size * sizeof(pix));
    hipMemcpy(d_vals, a, size * sizeof(pix), hipMemcpyHostToDevice);
    ThrustSortStruct(a, d_vals, size);
    hipFree(d_vals);
}

void ThrustSortStruct(pix* h_p, pix* d_p, int size)
{
    auto start = std::chrono::high_resolution_clock::now();
    thrust::device_ptr<pix> td_vals(d_p);
    thrust::sort(td_vals, td_vals+size, d_compPix());
    auto stop = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
    std::cout << "launch thrust sort struct call time: " << duration.count() << "\n";
    hipMemcpy(h_p, d_p, size * sizeof(unsigned int), hipMemcpyDeviceToHost);
}


// working qpix sorting functions, assume that the ion memory is already allocated, and we just
// need to make the memory a thrust vector
void ThrustQSort(Qpix::ION* qion, int size)
{
    thrust::device_ptr<Qpix::ION> d_qion_ptr(qion);
    thrust::sort(d_qion_ptr, d_qion_ptr+size, compIon());
}

thrust::device_vector<Pixel_Current> ThrustQMerge(Qpix::ION* qion, int size)
{
    // hard code 10 ns, for now
    double binSize = 10e-9;
    thrust::device_vector<Pixel_Current> d_pixel_current(size);

    // thrust::device_vector<Pixel_Current> d_pixel_current = thrust::device_malloc<Pixel_Current>(size);
    thrust::device_ptr<Qpix::ION> d_qion_ptr(qion);
    thrust::transform(d_qion_ptr, d_qion_ptr+size, d_pixel_current.begin(), single_pixelCurrent(binSize));
    thrust::inclusive_scan(d_pixel_current.begin(), d_pixel_current.end(), d_pixel_current.begin(), pixelCurrentSum());

    // down select the highest unique values in this list! 
    // go in reverse since the maximum values are on the 'right'
    auto d_uniq_pixel_current = thrust::unique(d_pixel_current.rbegin(), d_pixel_current.rend(), nextPixelTime());
    int uniq_length = thrust::distance(d_pixel_current.rbegin(), d_uniq_pixel_current);

    thrust::device_vector<Pixel_Current> uniq_data(uniq_length);
    thrust::copy(d_pixel_current.rbegin(), d_uniq_pixel_current, uniq_data.begin());

    return uniq_data;
}


