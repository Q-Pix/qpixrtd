#include "hip/hip_runtime.h"
#include "RTDCuda.h"

#include <iostream>
#include <numeric>
#include <iomanip>

#include <stdlib.h>

// #define MAX_DEPTH 2048
#define INSERTION_SORT 32

// reimplemented from Qpix::Functions
__device__ inline int ID_Encoder(const int& pix_x, const int& pix_y)
{
    return (int)(pix_x*10000+pix_y);
}


__global__ void makeQPixIons(double* start_x, double* step_x, double *start_y, double *step_y,
                             double* start_z, double* step_z, double *start_t, double *step_t, 
                             Qpix::ION * dest, int* Pix_ID, int* Trk_ID, double* time, int* count, 
                             int* hit_id, int size, int nHits, Qpix::Qpix_Paramaters qp_params,
                             hiprandState* state)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid < size) {

        hiprandState localState = state[tid]; 
        double rand_x = hiprand_normal_double(&localState);
        double rand_y = hiprand_normal_double(&localState);
        double rand_z = hiprand_normal_double(&localState);

        // each thread looks for it's position within nHits
        for(int i=0; i<nHits; ++i){
            if(count[i] >= tid + 1){
                dest[tid].x = start_x[i] - step_x[i] * (tid + 1 - count[i]);
                dest[tid].y = start_y[i] - step_y[i] * (tid + 1 - count[i]);
                dest[tid].z = start_z[i] - step_z[i] * (tid + 1 - count[i]);
                dest[tid].t = start_t[i] - step_t[i] * (tid + 1 - count[i]);

                Trk_ID[tid] = hit_id[i]; // marks which parent caused this ion

                double T_drift = dest[tid].z / qp_params.E_vel;
                // diffuse the electrons position
                double sigma_T = sqrt(2*qp_params.DiffusionT*T_drift);
                double sigma_L = sqrt(2*qp_params.DiffusionL*T_drift);

                double px = dest[tid].x + sigma_T * rand_x; 
                double py = dest[tid].y + sigma_T * rand_y; 
                double pz = rand_z + sigma_L * rand_z; 

                // convert the electrons x,y to a pixel index
                int Pix_Xloc = (int) ceil(px / qp_params.Pix_Size);
                int Pix_Yloc = (int) ceil(py / qp_params.Pix_Size);

                Pix_ID[tid] = ID_Encoder(Pix_Xloc, Pix_Yloc);
                time[tid] = dest[tid].t + ( pz / qp_params.E_vel );

                return;
            }
        }
        dest[tid].x = -41;
        dest[tid].y = -41;
        dest[tid].z = -41;
        dest[tid].t = -41;
        return;
    }

};

extern "C" std::vector<Pixel_Current> 
Launch_Make_QPixIons(double* start_x, double* step_x, double *start_y, double *step_y, 
                     double* start_z, double* step_z, double *start_t, double *step_t, 
                     Qpix::ION * dest_ions, int* con, int* hit_id, int size, int nHits,
                     Qpix::Qpix_Paramaters qp_params, int seed)
{
    double *d_start_x, *d_step_x;
    double *d_start_y, *d_step_y;
    double *d_start_z, *d_step_z;
    double *d_start_t, *d_step_t;

    // storage IONs for x, y, z, t
    Qpix::ION *d_qpion;

    // storage electrons, but raw values not a struct
    int *d_Pix_ID, *d_Trk_ID;
    double *d_time;

    int *d_con, *d_hit_id;

    int blocksPerGrid = (size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    // random number generation
    hiprandState *d_devStates;
    hipMalloc((void **)&d_devStates, size * sizeof(hiprandState));
    setup_normal_kernel<<<blocksPerGrid, THREADS_PER_BLOCK>>>(d_devStates, size, seed);

    // Allocate device memory for ION destination
    auto start = std::chrono::high_resolution_clock::now();
    auto err = hipMalloc(&d_qpion, size * sizeof(Qpix::ION));
    err = hipMalloc(&d_Pix_ID, size * sizeof(int));
    err = hipMalloc(&d_Trk_ID, size * sizeof(int));
    err = hipMalloc(&d_time, size * sizeof(double));
    auto stop = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
    std::cout << "rand: " << size << " @ random setup time: " << duration.count() << "\n";
    if(err != 0){std::cout << "error.\n"; exit(-1);};

    hipMalloc(&d_con, nHits * sizeof(int));
    hipMalloc(&d_hit_id, nHits * sizeof(int));
    hipMalloc(&d_start_x, nHits * sizeof(double));
    hipMalloc(&d_step_x, nHits * sizeof(double));
    hipMalloc(&d_start_y, nHits * sizeof(double));
    hipMalloc(&d_step_y, nHits * sizeof(double));
    hipMalloc(&d_start_z, nHits * sizeof(double));
    hipMalloc(&d_step_z, nHits * sizeof(double));
    hipMalloc(&d_start_t, nHits * sizeof(double));
    hipMalloc(&d_step_t, nHits * sizeof(double));

    // Copy input data from host to device
    hipMemcpy(d_con, con, nHits * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_hit_id, hit_id, nHits * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_start_x, start_x, nHits * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_step_x, step_x, nHits * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_start_y, start_y, nHits * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_step_y, step_y, nHits * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_start_z, start_z, nHits * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_step_z, step_z, nHits * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_start_t, start_t, nHits * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_step_t, step_t, nHits * sizeof(double), hipMemcpyHostToDevice);

    // Launch the working kernel
    start = std::chrono::high_resolution_clock::now();
    makeQPixIons<<<blocksPerGrid, THREADS_PER_BLOCK>>>(d_start_x, d_step_x, d_start_y, d_step_y, 
                                                       d_start_z, d_step_z, d_start_t, d_step_t, 
                                                       d_qpion, d_Pix_ID, d_Trk_ID, d_time,
                                                       d_con, d_hit_id, size, nHits,
                                                       qp_params, d_devStates);
    hipDeviceSynchronize();
    stop = std::chrono::high_resolution_clock::now();
    duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
    std::cout << "make: " << size << " @ make time: " << duration.count() << "\n";

    // sort the IONs on the GPU before copying back
    start = std::chrono::high_resolution_clock::now();
    ThrustQSort(d_Pix_ID, d_Trk_ID, d_time, d_con, nHits, size);
    hipDeviceSynchronize();
    stop = std::chrono::high_resolution_clock::now();
    duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
    std::cout << "size: " << size << " @ Thrust Q Sort Time: " << duration.count() << "\n";

    // merge and count the relevant time steps
    start = std::chrono::high_resolution_clock::now();
    auto d_pixel_current = ThrustQMerge(d_Pix_ID, d_Trk_ID, d_time, d_con, size); // return a device_vector of pixel_current
    stop = std::chrono::high_resolution_clock::now();
    duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
    std::cout << "size: " << size << " @ Thrust Q Merge Time: " << duration.count() << "\n";
    std::vector<Pixel_Current> hv_pixel_current(d_pixel_current.begin(), d_pixel_current.end());
    int totalNelec = 0;
    for(unsigned long int i=0; i<hv_pixel_current.size(); ++i){
      totalNelec += hv_pixel_current[i].nElec;
    }
    if(totalNelec != size){
      std::cout << "WARNING!!! electron count mismatch" <<
                   "total nelec: " << totalNelec << 
                   " total size: " << size << "\n";
    }
    
    // Copy the result from device to host
    start = std::chrono::high_resolution_clock::now();
    hipMemcpy(dest_ions, d_qpion, size * sizeof(Qpix::ION), hipMemcpyDeviceToHost);
    stop = std::chrono::high_resolution_clock::now();
    duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
    std::cout << "size: " << size << " @ copy time: " << duration.count() << "\n";

    // Free device memory
    hipFree(d_devStates);

    hipFree(d_start_x);
    hipFree(d_step_x);
    hipFree(d_qpion);
    hipFree(d_Pix_ID);
    hipFree(d_Trk_ID);
    hipFree(d_time);
    hipFree(d_start_y);
    hipFree(d_step_y);
    hipFree(d_start_z);
    hipFree(d_step_z);
    hipFree(d_start_t);
    hipFree(d_step_t);
    hipFree(d_con);
    hipFree(d_hit_id);

    return hv_pixel_current;
};

extern "C" void
Launch_Make_QResets(std::vector<Pixel_Current>& vpc, int* pid, double* resets,
                    std::vector<std::vector<int>>& trk_weights)
{
 std::cout << "building.\n";
};


/* modified from: https://docs.nvidia.com/cuda/hiprand/device-api-overview.html */
__global__ void setup_normal_kernel(hiprandState* state,
                                    int nElectrons, int seed)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    /* Each thread uses 3 different dimensions */
    // hiprand_init(0, id, 0, &state[id]);
    // hiprand(&state[id]);
    if(id < nElectrons){
        hiprand_init(seed, 0, 0, &state[id]);
    }
}

// prototyping the sort function -> not nearly as fast as thrust::sort
extern "C" void Launch_QuickSort(unsigned int* h_input_keys, unsigned int* h_output_keys, const int size, const int max_depth)
{
    std::cout << "kernel launch from host with size: " << size << "\n";
    unsigned int *d_input_keys;
    hipMalloc((void**)&d_input_keys, size * sizeof(unsigned int));

    hipMemcpy(d_input_keys, h_input_keys, size * sizeof(unsigned int), hipMemcpyHostToDevice);

    // Launch the working kernel
    cdp_simple_quicksort<<<1, 1>>>(d_input_keys, 0, size-1, 0, max_depth);

    hipMemcpy(h_output_keys, d_input_keys, size * sizeof(unsigned int), hipMemcpyDeviceToHost);

    hipFree(d_input_keys);
}

/* Below working examples taking from cuda-samples */
////////////////////////////////////////////////////////////////////////////////
// Selection sort used when depth gets too big or the number of elements drops
// below a threshold.
////////////////////////////////////////////////////////////////////////////////
__device__ void selection_sort(unsigned int *data, int left, int right) {
  for (int i = left; i <= right; ++i) {
    unsigned min_val = data[i];
    int min_idx = i;

    // Find the smallest value in the range [left, right].
    for (int j = i + 1; j <= right; ++j) {
      unsigned val_j = data[j];

      if (val_j < min_val) {
        min_idx = j;
        min_val = val_j;
      }
    }

    // Swap the values.
    if (i != min_idx) {
      data[min_idx] = data[i];
      data[i] = min_val;
    }
  }
}

////////////////////////////////////////////////////////////////////////////////
// Very basic quicksort algorithm, recursively launching the next level.
////////////////////////////////////////////////////////////////////////////////
__global__ void cdp_simple_quicksort(unsigned int *data, int left, int right,
                                     int depth, const int max_depth) {
  // If we're too deep or there are few elements left, we use an insertion
  // sort...
  if (depth >= max_depth || right - left <= INSERTION_SORT) {
    selection_sort(data, left, right);
    return;
  }

  unsigned int *lptr = data + left;
  unsigned int *rptr = data + right;
  unsigned int pivot = data[(left + right) / 2];

  // Do the partitioning.
  while (lptr <= rptr) {
    // Find the next left- and right-hand values to swap
    unsigned int lval = *lptr;
    unsigned int rval = *rptr;

    // Move the left pointer as long as the pointed element is smaller than the
    // pivot.
    while (lval < pivot) {
      lptr++;
      lval = *lptr;
    }

    // Move the right pointer as long as the pointed element is larger than the
    // pivot.
    while (rval > pivot) {
      rptr--;
      rval = *rptr;
    }

    // If the swap points are valid, do the swap!
    if (lptr <= rptr) {
      *lptr++ = rval;
      *rptr-- = lval;
    }
  }

  // Now the recursive part
  int nright = rptr - data;
  int nleft = lptr - data;

  // Launch a new block to sort the left part.
  if (left < (rptr - data)) {
    hipStream_t s;
    hipStreamCreateWithFlags(&s, hipStreamNonBlocking);
    cdp_simple_quicksort<<<1, 1, 0, s>>>(data, left, nright, depth + 1, max_depth);
    hipStreamDestroy(s);
  }

  // Launch a new block to sort the right part.
  if ((lptr - data) < right) {
    hipStream_t s1;
    hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
    cdp_simple_quicksort<<<1, 1, 0, s1>>>(data, nleft, right, depth + 1, max_depth);
    hipStreamDestroy(s1);
  }
}