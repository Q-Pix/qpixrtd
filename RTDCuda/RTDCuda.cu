#include "hip/hip_runtime.h"
#include "RTDCuda.h"

#include <iostream>
#include <numeric>
#include <iomanip>

#include <stdlib.h>


__global__ void addArrays(int* a, int* b, int* c, int size) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid < size) {
        c[tid] = a[tid] + b[tid];
    }
}

extern "C" void launch_add_arrays(int* a, int* b, int* c, int size) {
    int* d_a, *d_b, *d_c;

    // Allocate device memory
    hipMalloc(&d_a, size * sizeof(int));
    hipMalloc(&d_b, size * sizeof(int));
    hipMalloc(&d_c, size * sizeof(int));

    // Copy input data from host to device
    hipMemcpy(d_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    std::cout << "Cuda Result..\n";
    for (int i = 0; i < size; i++) {
        std::cout << "a: " << a[i] << " " << ", b: " << b[i] << "\n";
    }
    std::cout << std::endl;

    // Launch the kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    std::cout << "adding blocks: " << blocksPerGrid << "\n";
    addArrays<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, size);

    // Copy the result from device to host
    hipMemcpy(c, d_c, size * sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "Cuda Result: ";
    for (int i = 0; i < size; i++) {
        std::cout << c[i] << std::endl;
    }
    std::cout << std::endl;

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}


__global__ void addDiffArrays(double* point, double* step, Qpix::ION * dest, int* count, int size, int nHits)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid < size) {
        for(int i=0; i<nHits; ++i){
            if(count[i] >= tid + 1){
                dest[tid].x = point[i] - step[i] * (tid + 1 - count[i]);
                return;
            }
        }
        dest[tid].x = -41;
        return;
    }

};

extern "C" void launch_add_diff_arrays(double* start, double* step, Qpix::ION * dest, int* con, int size, int nHits)
{
    double* d_a, *d_b;
    Qpix::ION *d_c;
    int* d_con;

    // Allocate device memory
    hipMalloc(&d_c, size * sizeof(Qpix::ION));

    hipMalloc(&d_a, nHits * sizeof(double));
    hipMalloc(&d_b, nHits * sizeof(double));
    hipMalloc(&d_con, nHits * sizeof(int));

    // Copy input data from host to device
    hipMemcpy(d_a, start, nHits * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, step, nHits * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_con, con, nHits * sizeof(int), hipMemcpyHostToDevice);

    // std::cout << "Cuda Diff Input..\n";
    // for (int i = 0; i < nHits; i++) {
    //     std::cout << "a: " << a[i] << " " << ", b: " << b[i] << ", con: " << con[i] <<"\n";
    // }
    // std::cout << std::endl;

    // Launch the kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    addDiffArrays<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, d_con, size, nHits);

    // Copy the result from device to host
    hipMemcpy(dest, d_c, size * sizeof(Qpix::ION), hipMemcpyDeviceToHost);

    // std::cout << std::fixed;
    // std::cout << std::setprecision(4);
    // std::cout << "Add Size of Diff Cuda Result: " << size << "\n";
    for (int i = 0; i < size; i++) {
        if(dest[i].x == 0)
            std::cout << "warning val at index: " << i << "\n";
    }
    // std::cout << std::endl;
    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_con);
};

__global__ void makeElectron(double* start_x, double* start_y, double* start_z, double* start_t,
                             double* step_x, double* step_y, double* step_z, double* step_t,
                              double* ionx,
                            //  Qpix::ION* ions,
                             int* nElectrons, int maxElec, int size_step)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid < maxElec) {
        for(int i=0; i<size_step; ++i){
            if(nElectrons[i] >= tid + 1){
                // dest[tid] = point[i] + step[i] * (tid + 1 - nElectrons[i]);
                ionx[tid] = start_x[i] - step_x[i] * (tid + 1 - nElectrons[i]);
                // ions[tid].x = start_x[i] - step_x[i] * (tid + 1 - nElectrons[i]);
                // ions[tid].y = start_y[i] - step_y[i] * (tid + 1 - nElectrons[i]);
                // ions[tid].z = start_z[i] - step_z[i] * (tid + 1 - nElectrons[i]);
                // ions[tid].t = start_t[i] - step_t[i] * (tid + 1 - nElectrons[i]);
                return;
            }
        }
        ionx[tid] = -41;
        // ions[tid].x = -41;
        // ions[tid].y = -42;
        // ions[tid].z = -42;
        // ions[tid].t = -42;
    }
}

extern "C" void makeElectrons(double* start_x, double* start_y, double* start_z, double* start_t,
                              double* step_x, double* step_y, double* step_z, double* step_t,
                              Qpix::ION* ions,
                              int* nElec, int nElecSize)
{
    // allocate memory for all of the electrons
    // Qpix::ION* d_ions;

    double* d_ions_x;
    int nElectrons = nElec[nElecSize -1];
    // hipMalloc((void**)&d_ions, nElectrons * sizeof(Qpix::ION));  
    hipMalloc((void**)&d_ions_x, nElectrons * sizeof(double));  

    // allocate and copy memory for the step information on each hit
    double *d_sx;
    double *d_stepx;
    // double *d_sx, *d_sy, *d_sz, *d_st;
    // double *d_stepx, *d_stepy, *d_stepz, *d_stept;
    int* d_nElec;

    // allocate the starting points
    hipMalloc((void**)&d_sx, nElecSize * sizeof(double));  
    // hipMalloc((void**)&d_sy, nElecSize * sizeof(double));  
    // hipMalloc((void**)&d_sz, nElecSize * sizeof(double));  
    // hipMalloc((void**)&d_st, nElecSize * sizeof(double));  
    hipMalloc((void**)&d_stepx, nElecSize * sizeof(double));  
    // hipMalloc((void**)&d_stepy, nElecSize * sizeof(double));  
    // hipMalloc((void**)&d_stepz, nElecSize * sizeof(double));  
    // hipMalloc((void**)&d_stept, nElecSize * sizeof(double));  
    hipMalloc((void**)&d_nElec, nElecSize * sizeof(int));  

    // copy the starting points
    hipMemcpy(start_x, d_sx, nElecSize * sizeof(double), hipMemcpyHostToDevice);
    // hipMemcpy(start_y, d_sy, nElecSize * sizeof(double), hipMemcpyHostToDevice);
    // hipMemcpy(start_z, d_sz, nElecSize * sizeof(double), hipMemcpyHostToDevice);
    // hipMemcpy(start_t, d_st, nElecSize * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(step_x, d_stepx, nElecSize * sizeof(double), hipMemcpyHostToDevice);
    // hipMemcpy(step_y, d_stepy, nElecSize * sizeof(double), hipMemcpyHostToDevice);
    // hipMemcpy(step_z, d_stepz, nElecSize * sizeof(double), hipMemcpyHostToDevice);
    // hipMemcpy(step_t, d_stept, nElecSize * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(nElec, d_nElec, nElecSize * sizeof(int), hipMemcpyHostToDevice);

    dim3 blockDim(256,1,1);
    dim3 gridDim((nElectrons + blockDim.x - 1) / blockDim.x, 1, 1);
    // std::cout << "producing n blocks: " << gridDim.x << "\n";
    // makeElectron<<<gridDim, blockDim>>>(d_sx, d_sy, d_sz, d_st,
    //                                     d_stepx, d_stepy, d_stepz, d_stept,
    //                                     d_ions_x,
    //                                     // d_ions,
    //                                     d_nElec, nElectrons, nElecSize);

    int threadsPerBlock = 256;
    int blocksPerGrid = (nElectrons + threadsPerBlock - 1) / threadsPerBlock;
    // addDiffArrays<<<blocksPerGrid, threadsPerBlock>>>(d_sx, d_stepx, d_ions_x, d_nElec, nElectrons, nElecSize);

    // hipMemcpy(ions, d_ions, nElectrons * sizeof(Qpix::ION), hipMemcpyDeviceToHost);
    std::vector<double> vx(nElectrons);
    hipMemcpy(vx.data(), d_ions_x, nElectrons * sizeof(double), hipMemcpyDeviceToHost);

    // if(ions[0].x != -41)
    if(vx[0] != -41)
        std::cout << "found n ions: " << nElectrons << ", electron.x: " << vx[0] << std::endl;

    // free bird
    // hipFree(d_ions);
    hipFree(d_ions_x);
    hipFree(d_sx);
    // hipFree(d_sy);
    // hipFree(d_sz);
    // hipFree(d_st);
    hipFree(d_stepx);
    // hipFree(d_stepy);
    // hipFree(d_stepz);
    // hipFree(d_stept);
    hipFree(d_nElec);
};